
#include <hip/hip_runtime.h>
//!nvcc -arch=sm_70 -o thread-and-block-idx 03-indices/01-thread-and-block-idx.cu -run


// set NThreads as a multiple of 32 for performance optimization of CPU instructions

// max threads per block are  1024

//cudaFree and   cudaMallocManaged(&a, size) instead of malloc and free


//make a directory of kernels


// profile with !nsys profile --stats=true ./iteratively-optimized-vector-add

// assign #StreamProcessors to #Blocks for perofrmance opt. (load balance)
